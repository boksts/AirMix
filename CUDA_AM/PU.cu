#include "hip/hip_runtime.h"
#pragma once

#include <math.h>
//#include <time.h>
#include  <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "MiniWrapForCuda.h"
#include <ctime>


#define  b 100.0f;

#define BLOCK_SIZE 32

using namespace std;

//������ ��� ���������� �������
class Time{
	hipEvent_t Tn, Tk;
	float time;
public:
	

	Time(){
		hipEventCreate(&Tn);
		hipEventCreate(&Tk);
	}
	~Time(){
		hipEventDestroy(Tn);
		hipEventDestroy(Tk);
	}
	void tn(){
		hipEventRecord(Tn, 0);
	}
	float tk(){
		hipEventRecord(Tk, 0);
		hipEventSynchronize(Tk);
		hipEventElapsedTime(&time, Tn, Tk);
		return time;
	}
};
	


//���������� ��������
__global__ void kernel_P(int X, int Y, int x0, int l, double *P, double *Ux, double *Uy,double tau,double h){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if (i<0 || j<0 || i>(X - 1) || j>(Y - 1))
		return;

	if ((i>0) && (j>0) && (i<(X - 1)) && (j<(Y - 1))){
		P[j*X + i] = P[j*X + i] - tau*100.0*(((Ux[(j + 1)*X + i + 1] + Ux[(j - 1)*X + i + 1]) - (Ux[(j - 1)*X + i - 1] + Ux[(j + 1)*X + i - 1]) +
			(Uy[(j + 1)*X + i - 1] + Uy[(j + 1)*X + i + 1]) - (Uy[(j - 1)*X + i - 1] + Uy[(j - 1)*X + i + 1])) / (4.0*h));


		P[i] = P[X + i];
		P[(Y - 1)*X + i] = P[(Y - 2)*X + i];
		P[j*X] = P[1 + j*X];
		P[j*X + (X - 1)] = P[j*X + (X - 2)];
	}

	P[j*X] = 2 * P[1 + j*X] - P[2 + j*X];
	P[j*X + (X - 1)] = 2 * P[j*X + (X - 2)] - P[j*X + (X - 3)];

	if ((i >= x0) && (i <= x0 + l))
		P[(Y - 1)*X + i] = 2 * P[(Y - 2)*X + i] - P[(Y - 3)*X + i];

}


//���������� ����� ���������
__global__ void kernel_U(int X, int Y, double *Uxn, double *Uyn, double *P, double *Ux, double *Uy,double tau,double h,double nuM,double ro){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if ((i<(X - 1)) && (j<(Y - 1)) && (i>0) && (j>0)){

		Uxn[j*X + i] = Ux[j*X + i] + tau*(
			-(Ux[j*X + i] + abs(Ux[j*X + i])) / 2.0 * (Ux[j*X + i] - Ux[j*X + i - 1]) / h
			- (Ux[j*X + i] - abs(Ux[j*X + i])) / 2.0 * (Ux[j*X + i + 1] - Ux[j*X + i]) / h
			- (Uy[j*X + i] + abs(Uy[j*X + i])) / 2.0 * (Ux[j*X + i] - Ux[(j - 1)*X + i]) / h
			- (Uy[j*X + i] - abs(Uy[j*X + i])) / 2.0 * (Ux[(j + 1)*X + i] - Ux[j*X + i]) / h
			- (P[(j + 1)*X + i + 1] + P[(j - 1)*X + i + 1] - P[(j + 1)*X + i - 1] - P[(j - 1)*X + i - 1]) / (4 * h*ro)
			+ nuM*(Ux[j*X + i + 1] + Ux[j*X + i - 1] + Ux[(j - 1)*X + i] + Ux[(j + 1)*X + i] - 4 * Ux[j*X + i]) / (h*h));

		Uyn[j*X + i] = Uy[j*X + i] + tau*(
			-(Ux[j*X + i] + abs(Ux[j*X + i])) / 2.0 * (Uy[j*X + i] - Uy[j*X + i - 1]) / h
			- (Ux[j*X + i] - abs(Ux[j*X + i])) / 2.0 * (Uy[j*X + i + 1] - Uy[j*X + i]) / h
			- (Uy[j*X + i] + abs(Uy[j*X + i])) / 2.0 * (Uy[j*X + i] - Uy[(j - 1)*X + i]) / h
			- (Uy[j*X + i] - abs(Uy[j*X + i])) / 2.0 * (Uy[(j + 1)*X + i] - Uy[j*X + i]) / h
			- (P[(j + 1)*X + i - 1] + P[(j + 1)*X + i + 1] - P[(j - 1)*X + i - 1] - P[(j - 1)*X + i + 1]) / (4 * h*ro)
			+ nuM*(Uy[j*X + i + 1] + Uy[j*X + i - 1] + Uy[(j - 1)*X + i] + Uy[(j + 1)*X + i] - 4 * Uy[j*X + i]) / (h*h));
	}

}

//����������������
__global__ void kernel_p(int X, int Y, double *Uxn, double *Uyn, double *Ux, double *Uy){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if ((i<(X - 1)) && (j<(Y - 1)) && (i>0) && (j>0)){
		Ux[j*X + i] = Uxn[j*X + i];
		Uy[j*X + i] = Uyn[j*X + i];
	}
}


double *UxDev = NULL, *UyDev = NULL, *UxnDev = NULL, *UynDev = NULL, *PDev = NULL;
int X, Y;
int x0, len;
double tau, h;
double nuM, ro;
int sizef;
Time* timer;
double fulltime;
//FILE *f;
int gridSizeX,gridSizeY;

double Compute(ComputeOnCUDA::PU::PressureCalcMethod pressureMethod, ComputeOnCUDA::PU::NavierStokesCalcMethod navierStokesMethod, double *Ux, double *Uy, double tmax) {
	double t = 0;

	//����������� ����� ������ � �������
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(gridSizeX, gridSizeY);

	//����������� �������� � ����� � ������ ����������
	hipMemcpy(UxDev, Ux, sizef, hipMemcpyHostToDevice);
	hipMemcpy(UyDev, Uy, sizef, hipMemcpyHostToDevice);

	do{
		//������ ���� ����������
		kernel_P <<<blocks, threads >>>(X, Y, x0, len, PDev, UxDev, UyDev,tau,h);
		kernel_U <<<blocks, threads >>>(X, Y, UxnDev, UynDev, PDev, UxDev, UyDev,tau,h,nuM,ro);
		kernel_p <<<blocks, threads >>>(X, Y, UxnDev, UynDev, UxDev, UyDev);
		t += tau;

	} while (t <= tmax);

	//������������� ���������� � �����
	hipDeviceSynchronize();

	//����������� �������� � ���������� � ������ �����
	hipMemcpy(Ux, UxDev, sizef, hipMemcpyDeviceToHost);
	hipMemcpy(Uy, UyDev, sizef, hipMemcpyDeviceToHost);

	fulltime=timer->tk();

	return fulltime/1000.0;

	/*
	for (int j = 0; j < Y; j++){
		for (int i = 0; i < X; i++)


			fprintf(f, "%8.3f ", Ux[j*X + i]);
		fprintf(f, "\n");
	}

	fprintf(f, "\n X=%d ,Y=%d ,tmax=%f ,h=%f ,x0=%d ,len=%d ,tau=%f ", X, Y, tmax, h, x0, len, tau);*/
}

void Constructor(double _tau, double _ro, double _nuM, int _x0, int _len, double _h, int _X, int _Y){
	tau = _tau;
	ro = _ro;
	nuM = _nuM;
	x0 = _x0;
	len = _len;
	h = _h;
	X = _X;
	Y = _Y;
	
	//f = fopen("res.txt", "w");
	double *P = new double[X*Y];//��������

	timer = new Time();

	//��������� �������
	for (int i = 0; i < X; i++)
		for (int j = 0; j < Y; j++) {
			P[j * X + i] = 0.0;
		}

	sizef = X*Y*sizeof(double);

	//����������� ������� �����
	gridSizeX = (X / BLOCK_SIZE) + ((X % BLOCK_SIZE) > 0 ? 1 : 0);
	gridSizeY = (Y / BLOCK_SIZE) + ((Y % BLOCK_SIZE) > 0 ? 1 : 0);

	//��������� ������ �� ����������
	hipMalloc((void**)&UxDev, sizef);
	hipMalloc((void**)&UxnDev, sizef);
	hipMalloc((void**)&UyDev, sizef);
	hipMalloc((void**)&UynDev, sizef);
	hipMalloc((void**)&PDev, sizef);

	//����� ������ ������� ����������
	timer->tn();

	hipMemcpy(PDev, P, sizef, hipMemcpyHostToDevice);
	//fprintf(f, "������ ��������\n");

//����� ������ ������� ����������
	//timer = time->tk();


	//����� ����������� � ����
	//fprintf(f,"time=%f",timer/1000);


	/*for (i = 0; i<X; i++){
		for (j = 0; j<Y; j++)
			fprintf(f, "%7.3f ", Uy[j*X + i]);
		fprintf(f, "\n");
	}
	fprintf(f, "\n\n");*/


	/*for (int j = 0; j < Y; j++){
		for (int i = 0; i < X; i++)


			fprintf(f, "%8.3f ", Ux[j*X+i]);
		fprintf(f, "\n");
	}

	fprintf(f, "\n X=%d ,Y=%d ,tmax=%f ,h=%f ,x0=%d ,len=%d ,tau=%f ", X, Y, tmax, h, x0, len, tau);*/


	//delete time;

}

void Destructor() {
	//fprintf(f, "������ �����������, ���� ������\n");
	//fclose(f);
	hipFree(UxDev);
	hipFree(UxnDev);
	hipFree(UyDev);
	hipFree(UynDev);
	hipFree(PDev);

}