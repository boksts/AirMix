#include "hip/hip_runtime.h"
#pragma once

#include <math.h>
#include  <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "MiniWrapForCuda.h"
#include <ctime>

#define  epsPsi 0.001f
#define _BLOCK_SIZE 32

#define a 0.1f
#define c 1.0f
#define g 9.8f

#define betta  0.003665f
#define tetta  1.85f

using namespace std;

// ������ ��� ���������� �������
class _Time{
	hipEvent_t Tn, Tk;
	float time;
public:
	_Time(){
		hipEventCreate(&Tn);
		hipEventCreate(&Tk);
	}
	~_Time(){
		hipEventDestroy(Tn);
		hipEventDestroy(Tk);
	}
	void tn(){
		hipEventRecord(Tn, 0);
	}
	float tk(){
		hipEventRecord(Tk, 0);
		hipEventSynchronize(Tk);
		hipEventElapsedTime(&time, Tn, Tk);
		return time;
	}
};

//��������� ����������� (������������� �����������)
__global__ void kernel_gelmgolca(int X, int Y, double *w, double *wn, double *psi, double *ux, double *uy, double*Temp, double h, double tau, double nuM){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if ((i<X) && (j<Y)){
		w[i] = -(psi[i + X] - psi[i]) / (h*h);
		w[i + (Y - 1)*X] = -(psi[i + (Y - 2)*X] - psi[i + (Y - 1)*X]) / (h*h);
		w[j*X] = 0;
		w[j*X + (X - 1)] = w[j*X + (X - 2)];
	}

	if ((i<(X - 1)) && (j<(Y - 1)) && (i>0) && (j>0)){
		float dux, duy;

		if (ux[j*X + i] < 0)
			dux = (w[j*X + i + 1] - w[j*X + i]) / h;
		else
			dux = (w[j*X + i] - w[j*X + i - 1]) / h;
		if (uy[j*X + i] < 0)
			duy = (w[(j + 1)*X + i] - w[j*X + i]) / h;
		else
			duy = (w[j*X + i] - w[(j - 1)*X + i]) / h;

		wn[j*X + i] = w[j*X + i] + tau*(-ux[j*X + i] * dux - uy[j*X + i] * duy + nuM*
			(w[j*X + i + 1] + w[j*X + i - 1] + w[(j + 1)*X + i] + w[(j - 1)*X + i] - 4 * w[j*X + i]) / (h*h)
			- g*betta*Temp[j*X + i]);
	}
}

//��������� �������� (����� ������� ����������)
__global__ void kernel_puasson(int X, int Y, double *psi, double *w, int *pr, double h, double *psin){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if ((i<(X - 1)) && (j<(Y - 1)) && (i>0) && (j>0)){
		pr[j*X + i] = 0;
		psin[j*X + i] = 0.25*(psi[j*X + i + 1] + psi[j*X + i - 1] +
			psi[(j + 1)*X + i] + psi[(j - 1)*X + i] + h*h*w[j*X + i]);

		if (fabs(psin[j*X + i] - psi[j*X + i]) >= epsPsi)
			pr[j*X + i] = 1;
	}

}

//���������� ���������
__global__ void kernel_skorosti(int X, int Y, double *psi, double *ux, double *uy, double h){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if ((i>0) && (j>0) && (i<(X - 1)) && (j<(Y - 1))){
		ux[j*X + i] = -(psi[(j + 1)*X + i + 1] + psi[(j + 1)*X + i - 1]
			- psi[(j - 1)*X + i + 1] - psi[(j - 1)*X + i - 1]) / (4 * h);

		uy[j*X + i] = (psi[(j + 1)*X + i + 1] - psi[(j + 1)*X + i - 1]
			+ psi[(j - 1)*X + i + 1] - psi[(j - 1)*X + i - 1]) / (4 * h);
	}

}

//����������������
__global__ void _kernel_pTemp(int X, int Y, int x0, int len, double *Temp, double *Tempn){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if ((i<(X - 1)) && (j<(Y - 1)) && (i>0) && (j>0)){
		Temp[j*X + i] = Tempn[j*X + i];

		if ((i < x0) || (i >= x0 + len))
			Temp[(Y - 1)*X + i] = Tempn[(Y - 1)*X + i];

		Temp[i] = Tempn[i];
	}
}

//����������������
__global__ void kernel_p(int X, int Y, double *psi, double *psin){
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y*blockDim.y + threadIdx.y + 1;
	if ((i<(X - 1)) && (j<(Y - 1)))
		psi[j*X + i] = psin[j*X + i];
}

//���������� �����������
__global__ void _kernel_temp(int X, int Y, int x0, int len, double *Ux, double *Uy, double *Temp, double *Tempn, double nuM, double h, double tau){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if ((i<(X - 1)) && (j<(Y - 1)) && (i>0) && (j>0)){
		Tempn[j*X + i] = Temp[j*X + i] + tau * (-(Ux[j*X + i] + abs(Ux[j*X + i])) / 2.0 * (Temp[j*X + i] - Temp[j*X + i - 1]) / h
			- (Ux[j*X + i] - abs(Ux[j*X + i])) / 2.0 * (Temp[j*X + i + 1] - Temp[j*X + i]) / h
			- (Uy[j*X + i] + abs(Uy[j*X + i])) / 2.0 * (Temp[j*X + i] - Temp[(j - 1)*X + i]) / h
			- (Uy[j*X + i] - abs(Uy[j*X + i])) / 2.0 * (Temp[(j + 1)*X + i] - Temp[j*X + i]) / h
			+ c*(nuM)* (Ux[j*X + i + 1] + Ux[j*X + i - 1] + Uy[(j + 1)*X + i] + Uy[(j - 1)*X + i] - 2 * Ux[j*X + i] - 2 * Uy[j*X + i]) /
			(h * h));
		
		Temp[j*X + X - 1] = Tempn[j*X + X - 2];
	}

	//����������� � �������
	if ((i<(X - 1)) && (i>0)){
		//�� ������� �����
		if ((i < x0) || (i >= x0 + len))
			Tempn[(Y - 1)*X + i] = Temp[(Y - 1)*X + i] +
			tau*a*a / (h*h)*
			(Temp[(Y - 1)*X + i + 1] + Temp[(Y - 1)*X + i - 1] + Temp[(Y - 2)*X + i] - 4 * Temp[(Y - 1)*X + i]);

		//�� ������� ������
		Tempn[i] = Temp[i] + tau*a*a / (h*h)*(Temp[i + 1, 0] + Temp[i - 1] + Temp[X + i] - 4 * Temp[i]);
	}
}

double *_UxDev = NULL, *_UyDev = NULL, *_UxnDev = NULL, *_UynDev = NULL, *wDev = NULL, *wnDev = NULL, *psiDev = NULL, *psinDev = NULL, *_TempDev = NULL, *_TempnDev;
int *prDev = NULL;
int _X, _Y;
int _x0, _len;
double _tau, _h;
double _nuM, _ro;
int _sizef, sizei;
int _gridSizeX, _gridSizeY;
_Time* _timer;

double ComputeWPsi(ComputeOnCUDA::WPsi::HelmholtzCalcMethod hcm, ComputeOnCUDA::TurbulenceModel tm, double *Ux, double *Uy, double *Temp, double tmax) {
	double t = 0;
	double fulltime;
	//����������� ����� ������ � �������
	dim3 threads(_BLOCK_SIZE, _BLOCK_SIZE);
	dim3 blocks(_gridSizeX, _gridSizeY);

	//����������� �������� � ����� � ������ ����������
	hipMemcpy(_UxDev, Ux, _sizef, hipMemcpyHostToDevice);
	hipMemcpy(_UyDev, Uy, _sizef, hipMemcpyHostToDevice);
	hipMemcpy(_TempDev, Temp, _sizef, hipMemcpyHostToDevice);

	bool flag = false;
	int *pr = NULL;
	pr = new int[_X*_Y];

	do{

		_kernel_temp << <blocks, threads >> >(_X, _Y, _x0, _len, _UxDev, _UyDev, _TempDev, _TempnDev, _nuM, _h, _tau);
		_kernel_pTemp << <blocks, threads >> >(_X, _Y, _x0, _len, _TempDev, _TempnDev);
		//������� ���� �����
		kernel_gelmgolca << <blocks, threads >> >(_X, _Y, wDev, wnDev, psiDev,
			_UxDev, _UyDev, _TempDev, _h, _tau, _nuM);
		kernel_p << <blocks, threads >> >(_X, _Y, wDev, wnDev);
		
		//������� ��������� �������� �� ���������� ��������
		do {
			flag = false;
			//������ ���� ����������
			kernel_puasson << <blocks, threads >> >(_X, _Y, psiDev, wDev, prDev, _h,psinDev);
			kernel_p << <blocks, threads >> >(_X, _Y, psiDev, psinDev);
			//������������� ���������� � �����
			hipDeviceSynchronize();
			//����������� �������� � ���������� � ������ �����
			hipMemcpy(pr, prDev, sizei, hipMemcpyDeviceToHost);
			for (int j = 1; j<_Y - 1; j++)
				for (int i = 1; i<_X - 1; i++)
					if (pr[j*_X + i] == 1){
						flag = true;
						j = _Y; i = _X;
					}

		} while (flag);

		//������� ��������
		kernel_skorosti << <blocks, threads >> >(_X, _Y, psiDev, _UxDev, _UyDev, _h);
		t += _tau;

	} while (t <= tmax);

	//������������� ���������� � �����
	hipDeviceSynchronize();

	//����������� �������� � ���������� � ������ �����
	hipMemcpy(Ux, _UxDev, _sizef, hipMemcpyDeviceToHost);
	hipMemcpy(Uy, _UyDev, _sizef, hipMemcpyDeviceToHost);
	hipMemcpy(Temp, _TempDev, _sizef, hipMemcpyDeviceToHost);

	fulltime = _timer->tk();
	return fulltime / 1000.0;

}

void ConstructorWPsi(double tau,  double nuM, int x0, int len, double h, int X, int Y, double *Ux, double *Uy){
	_tau = tau;
	_nuM = nuM;
	_x0 = x0;
	_len = len;
	_h = h;
	_X = X;
	_Y = Y;

	double *psi = new double[X*Y];//������� ����
	double *w = new double[X*Y];//������� ����
	_sizef = X*Y*sizeof(double);
	sizei = X*Y*sizeof(int);
	_timer = new _Time();

	//��������� �������
	for (int i = 0; i < X; i++)
		for (int j = 0; j < Y; j++) {
			psi[j * X + i] = 0.0;
			w[j * X + i] = 0.0;
		}
	//������� ���� �� ��������
	for (int i = X - 2; i >= 0; i--){
		if (i > x0 + len)
			psi[i + (Y - 1)*X] = 0.0;
		if ((i >= x0) && (i <= x0 + len))
			psi[i + (Y - 1)*X] = psi[i + (Y - 1)*X + 1] + fabs(Uy[i + (Y - 1)*X]) * h;
		if (i < x0)
			psi[i + (Y - 1)*X] = psi[i + (Y - 1)*X + 1];
	}

	for (int j = Y - 2; j >= 0; j--)
		psi[j*X] = psi[(j + 1)*X] + Ux[j*X] * h;

	for (int i = 1; i<X; i++)
		psi[i] = psi[i - 1];

	for (int j = Y - 2; j >= 0; j--)
		psi[j*X + (X - 1)] = psi[(j + 1)*X + (X - 1)] + Ux[j*X + (X - 1)] * h;
	
	//����������� ������� �����
	_gridSizeX = (X / _BLOCK_SIZE) + ((X % _BLOCK_SIZE) > 0 ? 1 : 0);
	_gridSizeY = (Y / _BLOCK_SIZE) + ((Y % _BLOCK_SIZE) > 0 ? 1 : 0);

	//��������� ������ �� ����������
	hipMalloc((void**)&_UxDev, _sizef);
	hipMalloc((void**)&_UxnDev, _sizef);
	hipMalloc((void**)&_UyDev, _sizef);
	hipMalloc((void**)&_UynDev, _sizef);
	hipMalloc((void**)&psiDev, _sizef);
	hipMalloc((void**)&psinDev, _sizef);
	hipMalloc((void**)&wDev, _sizef);
	hipMalloc((void**)&wnDev, _sizef);
	hipMalloc((void**)&prDev, sizei);
	hipMalloc((void**)&_TempDev, _sizef);
	hipMalloc((void**)&_TempnDev, _sizef);

	//����� ������ ������� ����������
	_timer->tn();

	hipMemcpy(wDev, w, _sizef, hipMemcpyHostToDevice);
	hipMemcpy(psiDev, psi, _sizef, hipMemcpyHostToDevice);
	
}

void DestructorWPsi() {
	hipFree(_UxDev);
	hipFree(_UxnDev);
	hipFree(_UyDev);
	hipFree(_UynDev);
	hipFree(wDev);
	hipFree(wnDev);
	hipFree(psiDev);
	hipFree(psinDev);
	hipFree(prDev);
	hipFree(_TempDev);
	hipFree(_TempnDev);
}