#include "hip/hip_runtime.h"
#pragma once

#include <math.h>
#include  <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "MiniWrapForCuda.h"
#include <ctime>

#define  epsPsi 0.001f

#define _BLOCK_SIZE 32

using namespace std;


// ������ ��� ���������� �������
class _Time{
	hipEvent_t Tn, Tk;
	float time;
public:


	_Time(){
		hipEventCreate(&Tn);
		hipEventCreate(&Tk);
	}
	~_Time(){
		hipEventDestroy(Tn);
		hipEventDestroy(Tk);
	}
	void tn(){
		hipEventRecord(Tn, 0);
	}
	float tk(){
		hipEventRecord(Tk, 0);
		hipEventSynchronize(Tk);
		hipEventElapsedTime(&time, Tn, Tk);
		return time;
	}
};

//��������� ����������� (������������� �����������)
__global__ void kernel_gelmgolca(int X, int Y, double *w, double *wn, double *psi, double *ux, double *uy, double h, double tau, double nuM){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if ((i<X) && (j<Y)){
		w[i] = -(psi[i + X] - psi[i]) / (h*h);
		w[i + (Y - 1)*X] = -(psi[i + (Y - 2)*X] - psi[i + (Y - 1)*X]) / (h*h);
		w[j*X] = 0;
		w[j*X + (X - 1)] = w[j*X + (X - 2)];
	}

	if ((i<(X - 1)) && (j<(Y - 1)) && (i>0) && (j>0)){
		float dux, duy;

		if (ux[j*X + i] < 0)
			dux = (w[j*X + i + 1] - w[j*X + i]) / h;
		else
			dux = (w[j*X + i] - w[j*X + i - 1]) / h;
		if (uy[j*X + i] < 0)
			duy = (w[(j + 1)*X + i] - w[j*X + i]) / h;
		else
			duy = (w[j*X + i] - w[(j - 1)*X + i]) / h;


		wn[j*X + i] = w[j*X + i] + tau*(-ux[j*X + i] * dux - uy[j*X + i] * duy + nuM*
			(w[j*X + i + 1] + w[j*X + i - 1] + w[(j + 1)*X + i] + w[(j - 1)*X + i] - 4 * w[j*X + i]) / (h*h));

	}
}


//��������� �������� (����� �����)
__global__ void kernel_puasson(int X, int Y, double *psi, double *w, double *psin, int *pr, double h){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if ((i<(X - 1)) && (j<(Y - 1)) && (i>0) && (j>0)){
		pr[j*X + i] = 0;
		psin[j*X + i] = 0.25*(psi[j*X + i + 1] + psi[j*X + i - 1] +
			psi[(j + 1)*X + i] + psi[(j - 1)*X + i] + h*h*w[j*X + i]);

		if (fabs(psin[j*X + i] - psi[j*X + i]) >= epsPsi)
			pr[j*X + i] = 1;
	}
}


//���������� ���������
__global__ void kernel_skorosti(int X, int Y, double *psi, double *ux, double *uy, double h){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if ((i>0) && (j>0) && (i<(X - 1)) && (j<(Y - 1))){
		ux[j*X + i] = -(psi[(j + 1)*X + i + 1] + psi[(j + 1)*X + i - 1]
			- psi[(j - 1)*X + i + 1] - psi[(j - 1)*X + i - 1]) / (4 * h);

		uy[j*X + i] = -(psi[(j + 1)*X + i + 1] - psi[(j + 1)*X + i - 1]
			+ psi[(j - 1)*X + i + 1] - psi[(j - 1)*X + i - 1]) / (4 * h);
	}

}


//����������������
__global__ void kernel_p(int X, int Y, double *psi, double *psin){
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y*blockDim.y + threadIdx.y + 1;
	if ((i<(X - 1)) && (j<(Y - 1)))
		psi[j*X + i] = psin[j*X + i];
}


double *_UxDev = NULL, *_UyDev = NULL, *_UxnDev = NULL, *_UynDev = NULL, *wDev = NULL, *wnDev = NULL, *psiDev = NULL, *psinDev = NULL;
int *prDev = NULL;
int _X, _Y;
int _x0, _len;
double _tau, _h;
double _nuM, _ro;
int _sizef, sizei;
double _fulltime;
int _gridSizeX, _gridSizeY;
_Time* _timer;


double ComputeWPsi(ComputeOnCUDA::WPsi::HelmholtzCalcMethod hcm, ComputeOnCUDA::TurbulenceModel tm,  double *Ux, double *Uy, double tmax) {
	double t = 0;
	
	//����������� ����� ������ � �������
	dim3 threads(_BLOCK_SIZE, _BLOCK_SIZE);
	dim3 blocks(_gridSizeX, _gridSizeY);

	//����������� �������� � ����� � ������ ����������
	hipMemcpy(_UxDev, Ux, _sizef, hipMemcpyHostToDevice);
	hipMemcpy(_UyDev, Uy, _sizef, hipMemcpyHostToDevice);

	bool flag = false;
	int *pr = NULL;
	pr = new int[_X*_Y];
	
	do{
		//������ ���� ����������
		kernel_gelmgolca << <blocks, threads >> >(_X, _Y, wDev, wnDev, psiDev,
			_UxDev, _UyDev, _h, _tau, _nuM);
		kernel_p << <blocks, threads >> >(_X, _Y, wDev, wnDev);
		
		//������� ��������� �������� �� ���������� ��������
		do {
			flag = false;
			//������ ���� ����������
			kernel_puasson << <blocks, threads >> >(_X, _Y, psiDev, wDev, psinDev, prDev, _h);
			kernel_p << <blocks, threads >> >(_X, _Y, psiDev, psinDev);
			//������������� ���������� � �����
			hipDeviceSynchronize();
			//����������� �������� � ���������� � ������ �����
			hipMemcpy(pr, prDev, sizei, hipMemcpyDeviceToHost);
			for (int j = 1; j<_Y - 1; j++)
				for (int i = 1; i<_X - 1; i++)
					if (pr[j*_X + i] == 1){
						flag = true;
						j = _Y; i = _X;
					}

		} while (flag);

		//������ ���� ����������
		kernel_skorosti << <blocks, threads >> >(_X, _Y, psiDev, _UxDev, _UyDev, _h);
		t += _tau;

	} while (t <= tmax);

	//������������� ���������� � �����
	hipDeviceSynchronize();

	//����������� �������� � ���������� � ������ �����
	hipMemcpy(Ux, _UxDev, _sizef, hipMemcpyDeviceToHost);
	hipMemcpy(Uy, _UyDev, _sizef, hipMemcpyDeviceToHost);

	 _fulltime = _timer->tk();
	 return _fulltime / 1000.0;
	
	
}

void ConstructorWPsi(double tau,  double nuM, int x0, int len, double h, int X, int Y, double *Ux, double *Uy){
	_tau = tau;
	_nuM = nuM;
	_x0 = x0;
	_len = len;
	_h = h;
	_X = X;
	_Y = Y;

	double *psi = new double[X*Y];//������� ����
	double *w = new double[X*Y];//������� ����
	_sizef = X*Y*sizeof(double);
	sizei = X*Y*sizeof(int);
	_timer = new _Time();

	//��������� �������
	for (int i = 0; i < X; i++)
		for (int j = 0; j < Y; j++) {
			psi[j * X + i] = 0.0;
			w[j * X + i] = 0.0;
		}
	//������� ���� �� ��������
	for (int i = X - 2; i >= 0; i--){
		if (i > x0 + len)
			psi[i + (Y - 1)*X] = 0.0;
		if ((i >= x0) && (i <= x0 + len))
			psi[i + (Y - 1)*X] = psi[i + (Y - 1)*X + 1] + Uy[i + (Y - 1)*X] * h;
		if (i < x0)
			psi[i + (Y - 1)*X] = psi[i + (Y - 1)*X + 1];
	}

	for (int j = Y - 2; j >= 0; j--)
		psi[j*X] = psi[(j + 1)*X] + Ux[j*X] * h;

	for (int i = 1; i<X; i++)
		psi[i] = psi[i - 1];


	for (int j = Y - 2; j >= 0; j--)
		psi[j*X + (X - 1)] = psi[(j + 1)*X + (X - 1)] + Ux[j*X + (X - 1)] * h;
	

	//����������� ������� �����
	_gridSizeX = (X / _BLOCK_SIZE) + ((X % _BLOCK_SIZE) > 0 ? 1 : 0);
	_gridSizeY = (Y / _BLOCK_SIZE) + ((Y % _BLOCK_SIZE) > 0 ? 1 : 0);

	//��������� ������ �� ����������
	hipMalloc((void**)&_UxDev, _sizef);
	hipMalloc((void**)&_UxnDev, _sizef);
	hipMalloc((void**)&_UyDev, _sizef);
	hipMalloc((void**)&_UynDev, _sizef);
	hipMalloc((void**)&psiDev, _sizef);
	hipMalloc((void**)&psinDev, _sizef);
	hipMalloc((void**)&wDev, _sizef);
	hipMalloc((void**)&wnDev, _sizef);
	hipMalloc((void**)&prDev, sizei);

	//����� ������ ������� ����������
	_timer->tn();

	hipMemcpy(wDev, w, _sizef, hipMemcpyHostToDevice);
	hipMemcpy(psiDev, psi, _sizef, hipMemcpyHostToDevice);
	
}

void DestructorWPsi() {
	hipFree(_UxDev);
	hipFree(_UxnDev);
	hipFree(_UyDev);
	hipFree(_UynDev);
	hipFree(wDev);
	hipFree(wnDev);
	hipFree(psiDev);
	hipFree(psinDev);
	hipFree(prDev);

}